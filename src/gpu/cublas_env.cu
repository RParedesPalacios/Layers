#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include <hipblas.h>
#include <stdio.h>

//
#include "../utils.h"
#include "execution.h"
#include "cublas_env.h"

//global variables for all program
extern bool isYes;
extern hipblasHandle_t p_cublas;
extern hipblasStatus_t status;
extern gpu_specs gpuI;
////

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
cublas_env::cublas_env()
{
alfa=1;
}
//Private:
//Check for cuBlas initialization
void cublas_env::check()
{
if (!isYes)
	{fprintf(stderr,"Cublas Enviroment not created\n");exit(1);}
}

//Check for cuBlas execution
void cublas_env::error()
{
if ( status!=  HIPBLAS_STATUS_SUCCESS)
{
   fprintf(stderr,"Problem in cuBlas execution getting: %s\n",_cudaGetErrorEnum(status));
   exit(1);

}
}

//Destroy handle
/*void cublas_env::destroyHandle()
{
check();
status = hipblasDestroy(p_cublas);
error();
isYes=0;
}

//Create handle
void cublas_env::createHandle()
{
status = hipblasCreate(&p_cublas);

error();

isYes=1;
}*/

//Set atomic mode operation or not
void cublas_env::setAtomicMode(bool flag)
{
check();
if (flag)
	status = hipblasSetAtomicsMode(p_cublas, HIPBLAS_ATOMICS_ALLOWED);
else
	status = hipblasSetAtomicsMode(p_cublas, HIPBLAS_ATOMICS_NOT_ALLOWED);

error();
}

//ALGEBRA OPERATOR

//Matrix to Matrix Operator

//Product
void cublas_env::matrixProduct(float* A, float* B, float* C,tensor_gpu_specs* sA, tensor_gpu_specs* sB, tensor_gpu_specs* sC,int acc,int tA, int tB)
{
//Cublas is inspired in blas. blas is fortran library an thus it uses column major order. Layers is written in c-style. However 
//it can be easily performed: C=A*B; A*B=(B'*A')'; C'=(B'*A'). So we need only to perform B*A as cublas implicitly transpose B and A and we read C in row major order
//so implicitly do transposition
beta=(float)acc;
hipblasOperation_t trA = HIPBLAS_OP_N;
hipblasOperation_t trB = HIPBLAS_OP_N;
ldA=sA->col;
ldB=sB->col;
ldC=sB->col;
m=sB->col;
n=sA->row;
k=sB->row;

if (tA==1)
{
	trA = HIPBLAS_OP_T;
	n=sA->col; 
}
if (tB==1)
{
	trB = HIPBLAS_OP_T;
   	m=sB->row;
	k=sB->col;
        ldC=sB->row;//leading dimension is only affected when transposing B (C=B*A). We have to take columns of the output C which is normally col, however because of trasposition the output col will be row.
}

status = hipblasSgemm(p_cublas,trB,trA,m,n,k,&alfa,B,ldB,A,ldA,&beta,C,ldC);

error();

}

//Elwise sum
void cublas_env::mat_ewsum_mat(float* A, float* B, float* C,tensor_gpu_specs* sA,tensor_gpu_specs* sB, int tA, int tB,float sca,float scb)
{

hipblasOperation_t trA = HIPBLAS_OP_N;
hipblasOperation_t trB = HIPBLAS_OP_N;

m=sA->col;
n=sB->row;
ldA=sA->col;
ldB=sB->col;
ldC=sA->col;

if (tA==1)
{
	trA = HIPBLAS_OP_T;
	m=sA->row;
	ldC=sA->row;
}
if (tB==1)
{
	trB = HIPBLAS_OP_T;
	n=sB->col;
	
}

alfa=sca;
beta=scb;
status = hipblasSgeam(p_cublas,trA,trB, m,n,&alfa,A,ldA,&beta,B,ldB,C,ldC);
error();

}


//Matrix to scalar operator: either sum or product sc*A or sc+A->only product, sum is not contemplated
void cublas_env::sc_prod_mat(float* mat_o, float* mat_i,float sc,tensor_gpu_specs* sC,int acc)
{
//inc=0 means overlap over C
//inc=1 means increments over C

beta=(float)acc;
m=sC->col;
n=sC->row;
ldA=sC->col;
ldB=sC->col;
ldC=sC->col;


status = hipblasSgeam(p_cublas, HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&sc,mat_i,ldA,&beta,mat_o,ldB,mat_o,ldC);

error();

}

//Scalar to vector operator
void cublas_env::sc_prod_vec(float* vec_o, float* vec_i,float sc,tensor_gpu_specs* sC,int acc)
{
//inc=0 means overlap over C
//inc=1 means increments over C
hipError_t error_cuda;
if (acc==0)
        {
	error_cuda=hipMemcpy(vec_o,vec_i,sC->row*sC->col*sizeof(float),hipMemcpyDeviceToDevice);	
	if(error_cuda!=hipSuccess)
   		{fprintf(stderr,"Error memory transfer %s\n",hipGetErrorString(error_cuda));exit(-1);}
	status=hipblasSscal(p_cublas, sC->row*sC->col, &sc, vec_o, 1);

         }

if (acc==1)
    {
    fprintf(stderr,"Not chequed cublas scalar vector product\n");
    exit(-1);
    status=hipblasSaxpy(p_cublas, sC->row*sC->col , &sc, vec_i, sizeof(float), vec_o, sizeof(float));

    }


error();
}

//vector to vector operator
void cublas_env::vec_sum_vec(float* vec_o, float* vec_i1,float* vec_i2,tensor_gpu_specs* sC,int acc,float sca, float scb)
{
//acc=0 means overlap over C
//acc=1 means increments over C

//first as cublas routine overwrite result we need to save input
hipError_t error_cuda;

if (acc==0)
        {
	//vec_o=vec_i2*scb;
	sc_prod_vec(vec_o,vec_i2,scb,sC,0);
        error_cuda=hipDeviceSynchronize();
	if(error_cuda!=hipSuccess)
   		{fprintf(stderr,"Error sync%s\n",hipGetErrorString(error_cuda));exit(-1);}

        //vec_o=sca*vec_i1+vec_o=sca*vec_i1+scb*vec_i2
        status=hipblasSaxpy(p_cublas, sC->row*sC->col , &sca, vec_i1, 1, vec_o, 1);
	error();
        }

if (acc==1)
        {
	//vec_o+=vec_i2*scb;
	sc_prod_vec(vec_o,vec_i2,scb,sC,1);
        error_cuda=hipDeviceSynchronize();
	if(error_cuda!=hipSuccess)
   		{fprintf(stderr,"Error sync%s\n",hipGetErrorString(error_cuda));exit(-1);}

	//vec_o+=vec_i1*sca;
	sc_prod_vec(vec_o,vec_i1,sca,sC,1);
        
        }


}


//Matrix transposition
void cublas_env::mat_transp(float* output, float* input, tensor_gpu_specs* sInp)
{
//cannot perform in place tranposition an thus we need an output array 
beta=0.0;
alfa=1.0;
n=sInp->col;
m=sInp->row;
ldA=n;
ldB=sInp->row;
ldC=sInp->row;
status = hipblasSgeam(p_cublas, HIPBLAS_OP_T,HIPBLAS_OP_N, m,n,&alfa,input,ldA,&beta,output,ldB,output,ldC);
error();

}



