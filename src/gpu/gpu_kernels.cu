#include "hip/hip_runtime.h"
//Developer: Juan Maroñas Molano 
//This file implements kernel cuda for Layers

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "gpu_kernels.h"

////////////////////////////////////////////
///////////NEURAL NETWORKS//////////////////
////////////////////////////////////////////

//ACTIVATION FUNCTIONS
//FORWARD

//Rectifier Linear Units
__global__ void ReLu(float* E, float* N, long int n_vals)
{
  int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
if (thread_id_x < n_vals)
  N[thread_id_x]=fmaxf(0.0,E[thread_id_x]);
}

//Exponential Linear Unit
//JUAN_FIX: make eficient without warping divergence
__global__ void ELU(float* E,float* N, float alfa ,long int n_vals)
{

  int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
  float a=0;
  if (thread_id_x < n_vals)
  {
	a=E[thread_id_x]; 
	if (a<0) //Warping divergence
		N[thread_id_x]=alfa*(expf(a)-1);
  }
}

//Sigmoid 
__global__ void Sigmoid(float* E,float* N ,long int n_vals)
{
  int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
  if (thread_id_x < n_vals)
     N[thread_id_x] = 1/(1+expf(-1*E[thread_id_x]));
}

//Softmax->implemented for not saturating
__global__ void Softmax(float* E,float* N,float* auxE ,long int sample_dim, long int n_vals)
{
//This way of programing allow no warp syncronization as we only need kernel optimization.
//Maybe use thrust library but could be tricky. Should study if it fit well with this problem. Think is not, we would need one thrust vector per row.
//On the other hand possibly implement reduction as in http://www.cuvilib.com/Reduction.pdf. Will need to call another function. This could be complecated also as we need to see which thread id implements softmax and which one computes maximum. For now simple approximation.
    float C_value=0;
    int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
    float maxCoef = E[thread_id_x*sample_dim];
    float actualCoef = 0;
    if (thread_id_x<n_vals)
    {
	    ///REALLY HIGH PROBABILITY OF BRANCH DIVERGENCE. 
            //Description: All of the threads that lie under one condition execute first (stalling the others) and then next. Assuming one clock cycle per operation we would need double time to execute one warp.
	    //Warping divergence: study reduction options for getting the maximum
	    #pragma omp parallel for
	    for (int cA = 1; cA < sample_dim; cA++)
		if (E[thread_id_x*sample_dim+cA] > maxCoef)
			 maxCoef=E[thread_id_x*sample_dim+cA];

	    //No warping divergence as all threads execute the same
	    #pragma omp parallel for
	    for (int cA = 0; cA < sample_dim; cA++)
		{
			actualCoef=expf(E[thread_id_x*sample_dim+cA]-maxCoef);
			auxE[thread_id_x*sample_dim+cA]=actualCoef;
                        C_value+=actualCoef;
		}
            #pragma omp parallel for
	    for (int cA=0; cA < sample_dim; cA++)
	       N[thread_id_x*sample_dim+cA]=auxE[thread_id_x*sample_dim+cA]/C_value;
    }
	
}
//BACKWARD (derivatives)

__global__ void dReLu(float *E,float* D, long int total_ops)
{

  int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
  if (thread_id_x<total_ops)
  {
        D[thread_id_x]=(float) E[thread_id_x] > 0;

  }
}

__global__ void dSigmoid(float *E,float* D, long int total_ops)
{
 
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;
  if (thread_id_x < total_ops)
  {
     float val = 1/(1+expf(-1*E[thread_id_x])); 
     D[thread_id_x] = val*(1-val);
  }

}

__global__ void dELU(float *E,float* N,float* D, long int total_ops)
{
int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;

  if (thread_id_x < total_ops)
  {
    bool val = E[thread_id_x] <= 0;
    D[thread_id_x]=val*N[thread_id_x]+1;
  }

}
//cost functions

__global__ void MC_loss(float* T, float* N,float* acc,int cols, long int total_ops, int* MC_err)
{

int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
int result_t=T[thread_id_x*cols];
float result_n=N[thread_id_x*cols]; //cross entropy is between 0 and 1

int row_max_t=0;
int row_max_n=0;
bool err;

int aux_t;
float aux_n;
if (thread_id_x < total_ops)
{
  //hipblasIsamax(p_cublas_gpu,cols,&(T[thread_id_x*cols]),1,&result_t);
  //hipblasIsamax(p_cublas_gpu,cols,&(N[thread_id_x*cols]),1,&result_n);
  for(int i=1;i<cols;i++)
  {
   aux_t=T[thread_id_x*cols+i];
   aux_n=N[thread_id_x*cols+i];

	if (aux_t>result_t)
	 {
		result_t=aux_t;
                row_max_t=i;
         }
        if (aux_n>result_n)
	 {
		result_n=aux_n;
                row_max_n=i;
         }
  }

  acc[thread_id_x]=row_max_t;//store for crossentropy
  atomicAdd(MC_err,(int)(row_max_t!=row_max_n));
}

}

__global__ void CE_loss(float* N, float* max_row_vec,float* CE_vec,int cols ,long int total_ops)
{

int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
  if (thread_id_x<total_ops)
  {

    float aux1;
    float aux2; 
    bool pos;
    bool neg;
    bool selector;
    int max_row=max_row_vec[thread_id_x];
    for(int j=0;j<cols;j++)
    {
        selector=j==max_row;

        pos=N[thread_id_x*cols+j]==0;
        neg=N[thread_id_x*cols+j]==1;

        aux1 = log(pos+!pos*N[thread_id_x*cols+j]);
        aux2 = log(neg+!neg*(1-N[thread_id_x*cols+j]));

        CE_vec[thread_id_x*cols+j]=selector*aux1+(1-selector)*aux2;
    }
  }
}


////////////////////////////////////////////
////////////////////////////////////////////
////////////////////////////////////////////

///////////////////////////////////////////
//////////////LINEAR ALGEBRA///////////////
///////////////////////////////////////////

//SCALAR MATRIX OPERATOR
//add scalar to matrix and store
__global__ void sc_add_mat(float* mat_o,float* mat_i, float sc, long int total_ops, int acc)
{
  //this function does not have memory races-> maybe optimized using shared memory
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (thread_id_x < total_ops)
  {
	if (acc == 0)//not warping divergence as all the threads will have acc set to same value
		mat_o[thread_id_x]=mat_i[thread_id_x]+sc;
	else if(acc==1)
		mat_o[thread_id_x]+=(mat_i[thread_id_x]+sc);
   }
}
////////////////////////////////////////////
////////////////////////////////////////////
////////////////////////////////////////////

///////////////////////////////////////////
//////////ELEMENT WISE OPERATOR////////////
///////////////////////////////////////////


//MATRIX MATRIX OPERATOR
//el wise product

__global__ void mat_ewprod_mat(float* mat_o,float* mat_i1, float* mat_i2,int acc ,long int total_ops)
{
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;
  if (thread_id_x < total_ops)
  {
      
	if (acc==0)//Accumulate the value
        {
		mat_o[thread_id_x]=mat_i1[thread_id_x]*mat_i2[thread_id_x];
        }
	else if(acc==1)
        {
		mat_o[thread_id_x]=mat_o[thread_id_x]+mat_i1[thread_id_x]*mat_i2[thread_id_x];
        }
   

  }
}
//tensor equal
__global__ void kern_tensor_equal(float* A, float* B, long int ops)
{
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;

  if(thread_id_x < ops)
	if(A[thread_id_x]!=B[thread_id_x])
		asm("trap;");

}

//MATRIX VECTOR OPERATOR
__global__ void mat_ewcol_vec(float* mat_o, float* mat, float* vec, long int ops, long int cols, int op,int acc,float sca, float scb)
{

  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
 if (acc==1)
 {
  if (thread_id_x < ops)
   if (op==0)
    mat_o[thread_id_x]=mat_o[thread_id_x]+ sca*mat[thread_id_x]+scb*vec[thread_id_x%cols];
   else if (op==1)
    mat_o[thread_id_x]= mat_o[thread_id_x]+mat[thread_id_x]*vec[thread_id_x%cols];
   else
    mat_o[thread_id_x]= mat_o[thread_id_x]+mat[thread_id_x]/vec[thread_id_x%cols];
 }

 else
 {
  if (thread_id_x < ops)
   if (op==0)
     mat_o[thread_id_x]=sca*mat[thread_id_x]+scb*vec[thread_id_x%cols];
   else if(op==1)
     mat_o[thread_id_x]=mat[thread_id_x]*vec[thread_id_x%cols];
   else
     mat_o[thread_id_x]=mat[thread_id_x]/vec[thread_id_x%cols];
 }

}

__global__ void mat_ewrow_vec(float* mat_o, float* mat, float* vec, long int ops, long int rows, int op,int acc,float sca, float scb)
{

 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
 if (acc==1)
 {
  if (thread_id_x < ops)
   if (op==0)
    mat_o[thread_id_x]=mat_o[thread_id_x]+ sca*mat[thread_id_x]+scb*vec[thread_id_x/rows];
   else if (op==1)
    mat_o[thread_id_x]= mat_o[thread_id_x]+mat[thread_id_x]*vec[thread_id_x/rows];
  else
    mat_o[thread_id_x]= mat_o[thread_id_x]+mat[thread_id_x]/vec[thread_id_x/rows];
 }
 else
 {
  if (thread_id_x < ops)
   if (op==0)
     mat_o[thread_id_x]=sca*mat[thread_id_x]+scb*vec[thread_id_x/rows];
   else if(op==1)
     mat_o[thread_id_x]=mat[thread_id_x]*vec[thread_id_x/rows];
   else
     mat_o[thread_id_x]=mat[thread_id_x]/vec[thread_id_x/rows];
 }
}
////////////////////////////////////////////
////////////////////////////////////////////
////////////////////////////////////////////


////////////////////////////////////////////
////////////MEMORY MANAGMENT////////////////
////////////////////////////////////////////
__global__ void tensor_set_value(float* A, float value, long int total_ops)
{
int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;

if (thread_id_x < total_ops)
	A[thread_id_x]=value;

}

////////////////////////////////////////////
//////////////INPLACE OPERATOR//////////////
////////////////////////////////////////////
__global__ void tensor_sqrt(float* o, float* i,int acc,long int total_ops)
{

int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
if (thread_id_x<total_ops)
{
  if (acc==0)
    o[thread_id_x]=sqrtf(i[thread_id_x]);
  else
    o[thread_id_x]+=sqrtf(i[thread_id_x]);
}
}

////////////////////////////////////////////
////////////REDUCTION OPERATOR//////////////
////////////////////////////////////////////
//JUAN_FIX: implement with reduction operators
__global__ void kernel_row_sum(float* I, float* O, int rows,int cols ,long int ops)
{

int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
int i=0;
 if (thread_id_x < ops)
   #pragma omp parallel for
   for(i=0;i<cols;i++)
        O[thread_id_x]+=I[cols*thread_id_x+i];

}

__global__ void kernel_col_sum(float* I, float* O, int rows,int cols ,long int ops)
{

int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
int i=0;
 if (thread_id_x < ops)
   #pragma omp parallel for
   for(i=0;i<rows;i++)
   	O[thread_id_x]+=I[thread_id_x+cols*i];
   
}
//sum all the elements in an array with a sum
//possibly bad implemented. However we do not expect lot of output units
__global__ void reduce_array_sum(float* array, long int ops, int cols,float* result)
{
extern __shared__ float arr_acc[];//this could not be enough and may call with recursion
__shared__ float accumulate_result[1];

int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
float ent=0;
arr_acc[thread_id_x]=0.0;

if(thread_id_x==0)
	accumulate_result[thread_id_x]=0.0;

__syncthreads();
if (thread_id_x<ops)  
{
	for (int i=0; i<cols;i++)
		ent-=array[thread_id_x*cols+i];

__syncthreads();
	arr_acc[thread_id_x]=ent;
__syncthreads();

}

if (thread_id_x==0)
{

        #pragma omp parallel for
	for (int i=0; i<ops;i++)
               accumulate_result[thread_id_x]+=arr_acc[thread_id_x+i];

result[thread_id_x]=accumulate_result[thread_id_x];//copy back to global memory from shared

}	
}
/*
for (unsigned int s=1; s< blockDim.x<s*=2)
{
  int index = 2*+thread_id_x;
  if (index < blockDim.x)
  {
    +=operator
  }

}
*/
//}


//////////////////////////////////////////////
///////////////RANDOM GENERATOR///////////////
//////////////////////////////////////////////
//Do this using device api
__global__ void drop_mask(float* rand_vec,float p,long int ops)
{
   int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
   if (thread_id_x<ops)
     rand_vec[thread_id_x]=rand_vec[thread_id_x]>p;

}

__global__ void add_noise_with_mask(float* nn_vector,float* rand_vec,float* mask,float noiser,long int ops)
{

   int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
   if (thread_id_x<ops)
     nn_vector[thread_id_x]+=(mask[thread_id_x]<noiser)*rand_vec[thread_id_x];

}




