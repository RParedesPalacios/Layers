#include "hip/hip_runtime.h"
//Developer: Juan Maroñas Molano 
//This file implements kernel cuda for Layers

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "gpu_kernels.h"

////////////////////////////////////////////
///////////NEURAL NETWORKS//////////////////
////////////////////////////////////////////

//ACTIVATION FUNCTIONS
//FORWARD

//Rectifier Linear Units
__global__ void ReLu(float* E, float* N, long int n_vals)
{
  int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
if (thread_id_x < n_vals)
  N[thread_id_x]=fmaxf(0.0,E[thread_id_x]);
}

//Exponential Linear Unit
//JUAN_FIX: make eficient without warping divergence
__global__ void ELU(float* E,float* N, float alfa ,long int n_vals)
{

  int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
  float a=0;
  if (thread_id_x < n_vals)
  {
	a=E[thread_id_x]; 
	if (a<0) //Warping divergence
		N[thread_id_x]=alfa*(expf(a)-1);
  }
}

//Sigmoid 
__global__ void Sigmoid(float* E,float* N ,long int n_vals)
{
  int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
  if (thread_id_x < n_vals)
     N[thread_id_x] = 1/(1+expf(-1*E[thread_id_x]));
}

//Softmax->implemented for not saturating
__global__ void Softmax(float* E,float* N,float* auxE ,long int sample_dim, long int n_vals)
{
//This way of programing allow no warp syncronization as we only need kernel optimization.
//Maybe use thrust library but could be tricky. Should study if it fit well with this problem. Think is not, we would need one thrust vector per row.
//On the other hand possibly implement reduction as in http://www.cuvilib.com/Reduction.pdf. Will need to call another function. This could be complecated also as we need to see which thread id implements softmax and which one computes maximum. For now simple approximation.
    float C_value=0;
    int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
    float maxCoef = E[thread_id_x*sample_dim];
    float actualCoef = 0;
    if (thread_id_x<n_vals)
    {
	    ///REALLY HIGH PROBABILITY OF BRANCH DIVERGENCE. 
            //Description: All of the threads that lie under one condition execute first (stalling the others) and then next. Assuming one clock cycle per operation we would need double time to execute one warp.
	    //Warping divergence: study reduction options for getting the maximum
	    #pragma omp parallel for
	    for (int cA = 1; cA < sample_dim; cA++)
		if (E[thread_id_x*sample_dim+cA] > maxCoef)
			 maxCoef=E[thread_id_x*sample_dim+cA];

	    //No warping divergence as all threads execute the same
	    #pragma omp parallel for
	    for (int cA = 0; cA < sample_dim; cA++)
		{
			actualCoef=expf(E[thread_id_x*sample_dim+cA]-maxCoef);
			auxE[thread_id_x*sample_dim+cA]=actualCoef;
                        C_value+=actualCoef;
		}
            #pragma omp parallel for
	    for (int cA=0; cA < sample_dim; cA++)
	       N[thread_id_x*sample_dim+cA]=auxE[thread_id_x*sample_dim+cA]/C_value;
    }
	
}
//BACKWARD (derivatives)

__global__ void dReLu(float *E,float* D, long int total_ops)
{

  int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
  if (thread_id_x<total_ops)
  {
        D[thread_id_x]=(float) E[thread_id_x] > 0;

  }
}

__global__ void dSigmoid(float *E,float* D, long int total_ops)
{
 
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;
  if (thread_id_x < total_ops)
  {
     float val = 1/(1+expf(-1*E[thread_id_x])); 
     D[thread_id_x] = val*(1-val);
  }

}

__global__ void dELU(float *E,float* N,float* D, long int total_ops)
{
int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;

  if (thread_id_x < total_ops)
  {
    bool val = E[thread_id_x] <= 0;
    D[thread_id_x]=val*N[thread_id_x]+1;
  }

}

////////////////////////////////////////////
////////////////////////////////////////////
////////////////////////////////////////////

///////////////////////////////////////////
//////////////LINEAR ALGEBRA///////////////
///////////////////////////////////////////

//SCALAR MATRIX OPERATOR
//add scalar to matrix and store
__global__ void sc_add_mat(float* mat_o,float* mat_i, float sc, long int total_ops, int acc)
{
  //this function does not have memory races-> maybe optimized using shared memory
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (thread_id_x < total_ops)
  {
	if (acc == 0)//not warping divergence as all the threads will have acc set to same value
		mat_o[thread_id_x]=mat_i[thread_id_x]+sc;
	else if(acc==1)
		mat_o[thread_id_x]+=(mat_i[thread_id_x]+sc);
   }
}
////////////////////////////////////////////
////////////////////////////////////////////
////////////////////////////////////////////

///////////////////////////////////////////
//////////ELEMENT WISE OPERATOR////////////
///////////////////////////////////////////


//MATRIX MATRIX OPERATOR
//el wise product

__global__ void mat_ewprod_mat(float* mat_o,float* mat_i1, float* mat_i2,int acc ,long int total_ops)
{
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;
  if (thread_id_x < total_ops)
  {
      
	if (acc==0)//Accumulate the value
        {
		mat_o[thread_id_x]=mat_i1[thread_id_x]*mat_i2[thread_id_x];
        }
	else if(acc==1)
        {
		mat_o[thread_id_x]=mat_o[thread_id_x]+mat_i1[thread_id_x]*mat_i2[thread_id_x];
        }
   

  }
}
//tensor equal
__global__ void kern_tensor_equal(float* A, float* B, long int ops)
{
  int thread_id_x = threadIdx.x + blockIdx.x * blockDim.x;

  if(thread_id_x < ops)
	if(A[thread_id_x]!=B[thread_id_x])
		asm("trap;");

}

//MATRIX VECTOR OPERATOR
__global__ void mat_ewcol_vec(float* mat, float* vec, long int ops, long int cols, int op)
{

  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
 
  if (thread_id_x < ops)
   if (op==0)
     mat[thread_id_x]+=vec[thread_id_x%cols];
   else
     mat[thread_id_x]*=vec[thread_id_x%cols];

}

////////////////////////////////////////////
////////////////////////////////////////////
////////////////////////////////////////////


////////////////////////////////////////////
////////////MEMORY MANAGMENT////////////////
////////////////////////////////////////////
__global__ void tensor_set_value(float* A, float value, long int total_ops)
{
int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;

if (thread_id_x < total_ops)
	A[thread_id_x]=value;

}

////////////////////////////////////////////
////////////SELF OPERATOR///////////////////
////////////////////////////////////////////
//JUAN_FIX: implement with reduction operators
__global__ void kernel_col_sum(float* I, float* O, int rows,int cols ,long int ops)
{

int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
int i=0;
if (thread_id_x < ops)
   #pragma omp parallel for
   for(i=0;i<rows;i++)
   	O[thread_id_x]+=I[thread_id_x+cols*i];
   
}

//////////////////////////////////////////////
///////////////RANDOM GENERATOR///////////////
//////////////////////////////////////////////
//Do this using device api
__global__ void drop_mask(float* rand_vec,float p,long int ops)
{
   int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
   if (thread_id_x<ops)
     rand_vec[thread_id_x]=rand_vec[thread_id_x]>p;

}

__global__ void add_noise_with_mask(float* nn_vector,float* rand_vec,float* mask,float noiser,long int ops)
{

   int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
   if (thread_id_x<ops)
     nn_vector[thread_id_x]+=(mask[thread_id_x]>noiser)*rand_vec[thread_id_x];

}




