#include <string.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

#include "../utils.h"
#include "../types.h"
#include "execution.h"
#include "gpu_env.h"
#include "gpu_kernels.h"

#define max_gpu_threads 10000000000

/////////////////CURAND ERROR/////////////////////////


static const char *_curandGetErrorEnum(hiprandStatus_t error)
{
    switch (error)
    {
        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";
            break;
        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";
            break;
        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";
            break;

        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";
            break;

        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";
            break;

        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";
            break;

        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";
            break;

        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
            break;
        default:
	    fprintf(stderr,"Not all hiprand errors here\n");
	    exit(-1);
    }

}

/////////////////////////////////////////////////////////////



gpu_env::gpu_env()
{

}
//error function for random numbers
void gpu_env::error_rand()
{
  if(rand_error!=HIPRAND_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuda random execution getting: %s\n",	_curandGetErrorEnum(rand_error));
     exit(-1);
  }

}
//error function for check internally
void gpu_env::error_f()
{
  if ( error!=hipSuccess)
  {
     fprintf(stderr,"Problem in cuda execution getting: %s\n",	hipGetErrorString(error));
     exit(-1);
  }

}

//////////////////////////////////////////
/////////////Memory Transfer//////////////
//////////////////////////////////////////

//JUAN_FIX: change for cuda malloc pitch
//make tensor in GPU and alloc memory
float* gpu_env::makeTensor( int a, int b, int c, int d)
{
  float* devicePointer;
  error=hipMalloc((void**)&devicePointer,a*b*c*d*sizeof(float));
  error_f();
  return devicePointer;
}
//destroy tensor
void gpu_env::destroyTensor(float* p)
{
  error=hipFree(p);
  error_f();
}
//transfer scalar array
void gpu_env::set_sc(float* gpu, float sc, tensor_gpu_specs* sp)
{
  if (gpu==NULL)
    {fprintf(stderr,"Correctly allocate tensor in GPU and CPU\n");exit(-1);}

   //use Memset if int value, for float we need a kernel
     //error=hipMemset(p,sc,size);
   dim3 dimBlock(sp->col);
   dim3 dimGrid(sp->row);
   long int ops=sp->batch*sp->row*sp->col*sp->featureMap;

   tensor_set_value<<<dimBlock,dimGrid>>>(gpu,sc,ops);
   error_f();
   error=hipDeviceSynchronize();
   error_f();
}
//JUAN_FIX:implement function which stack all the device pointer to dealloc when error happens.
//transfer array to array
void gpu_env::copy_data(float* cpu, float* gpu,tr_type t,size_t size)
{
  if (cpu==NULL || gpu==NULL)
	{fprintf(stderr,"Correctly allocate tensor in GPU and CPU\n");exit(-1);}

  if (t==TOGPU)
    error=hipMemcpy(gpu,cpu,size,hipMemcpyHostToDevice);
  else if (t==FROMGPU)
{
    error=hipMemcpy(cpu,gpu,size,hipMemcpyDeviceToHost);
}
  else
    {fprintf(stderr,"NOT IMPLEMENTED");exit(-1);} 

  error_f();
  error=hipDeviceSynchronize();
  error_f();
}

/////////////////////////////////////
/////////Neural Networks/////////////
/////////////////////////////////////

//activation functions
void gpu_env::activation(float* E,float* N, int f,tensor_gpu_specs* sp)
{
if (E==NULL || N==NULL)
	{fprintf(stderr,"Fill data in pointer.\n");exit(-1);}
dim3 dimBlock;
dim3 dimGrid;
if (f==ACT_SOF)
{
 dimBlock.x=sp->row;
 dimGrid.x=1;
}
else
{
 dimBlock.x=sp->col;
 dimGrid.x=sp->row;
}
long int ops = sp->col*sp->row;
long int sample_dim=sp->col;

double alfa=1;
switch(f)
{
  case ACT_RLU:
	ReLu<<<dimBlock,dimGrid>>>(E,N,ops);
	break;
  case ACT_ELU:
	ELU<<<dimBlock,dimGrid>>>(E,N,alfa,ops);
	break;
  case ACT_SOF:
       {
        ops=sp->row;
        float* auxE = makeTensor(sp->col,sp->row);
	Softmax<<<dimBlock,dimGrid>>>(E,N,auxE,sample_dim,ops);
	break;
       }
  case ACT_SIG:
	Sigmoid<<<dimBlock,dimGrid>>>(E,N,ops);
	break;
  case ACT_LIN:
        //N=E;
        error = hipMemcpy(N,E,sp->batch*sp->featureMap*sp->col*sp->row*sizeof(float),hipMemcpyDeviceToDevice);
        break;
  default:
	fprintf(stderr,"Activation function not implemented\n");
	exit(-1);
}
error_f();
error=hipDeviceSynchronize();
error_f();
}
//derivative of activation functions
void gpu_env::dactivation(float* E, float* N,float* D, int f,tensor_gpu_specs* sp)
{
if (E==NULL || N==NULL || D==NULL)
	{fprintf(stderr,"Fill data in pointer.\n");exit(-1);}
dim3 dimBlock(sp->col);
dim3 dimGrid(sp->row);
long int ops = 10000;
long int sample_dim=sp->col*sp->row;

switch(f)
{
  case ACT_RLU:
	dReLu<<<dimBlock,dimGrid>>>(E,D,ops);
	break;
  case ACT_ELU:
	dELU<<<dimBlock,dimGrid>>>(E,N,D,ops);
	break;
  case ACT_SIG:
	dSigmoid<<<dimBlock,dimGrid>>>(E,D,ops);
	break;
  case ACT_LIN:
       set_sc(D,1.0,sp);      
       break;
  default:
	fprintf(stderr,"Activation function not implemented\n");
	exit(-1);
}
error_f();
error=hipDeviceSynchronize();
error_f();

}
/////////////////////////////////////////
///////////////gpu parsing///////////////
////////////////////////////////////////
void gpu_env::gpu_info(int selected_gpu)
{

int nDevices;
hipGetDeviceCount(&nDevices);

if (selected_gpu>nDevices)
 {
  fprintf(stderr,"Error. GPU %d not available. Number of available GPU is %d. Further information running nvidia-smi\n",selected_gpu,nDevices);
  exit(-1);	
 }

fprintf(stderr,"Selecting GPU device %d\n",selected_gpu);
hipSetDevice(selected_gpu);

hipDeviceProp_t prop;
hipGetDeviceProperties(&prop,selected_gpu);

fprintf(stderr,"Layers is running on GPU %s\n",prop.name);

gpuI.warpSize=prop.warpSize;
gpuI.maxThreadPerBlock=prop.maxThreadsPerBlock;

}


///////////////////////////////////////////
///////////////Linear Albegra//////////////
///////////////////////////////////////////

//MATRIX TO MATRIX OPERATIONS

//matrix operator
//Perform matrix produc c=a*b
//float pointer are matrix
//tensor_gpu_spcs matrix specifications
//tA and tB are for transposing matrix
// acc{0,1}->whether to accumulate result in C (1) or set (0)
void gpu_env::matMul(float* a,float* b,float* c, tensor_gpu_specs* sA,tensor_gpu_specs* sB, tensor_gpu_specs* sC, int acc, int tA, int tB)
{
//for the moment only available cublas for operation
	cublas.matrixProduct(a,b,c,sA,sB,sC,acc,tA,tB);
	error=hipDeviceSynchronize();
	error_f();

}

//SCALAR VECTOR OPERATOR
void gpu_env::scVec(float* vec_o,float* vec_i, tensor_gpu_specs* sA, float sc ,int op,int acc)
{
//gpuI has info about GPU for kernel dimensions

if (op==0)
{
	dim3 dimBlock(sA->row);//maybe move this to declaration
	dim3 dimGrid(sA->col);
	long int ops = sA->row*sA->col;
	//add -> no blas operation (we could use same as product buth we should fill B in gpu and that is shit. Home made kernel)
	sc_add_mat<<<dimBlock,dimGrid>>>(vec_o,vec_i,sc,ops,acc);//could be same as matrix scalar operator because we treat as float*

	error_f();
	error=hipDeviceSynchronize();
	error_f();
}
else if (op==1)
{
	//prod -> use same as matrix + matrix just matrix B is 0
        if (vec_o==NULL || vec_i==NULL)
		{printf("Error\n");exit(-1);} 
	error=hipMemcpy(vec_o,vec_i,sA->row*sA->col*sizeof(float),hipMemcpyDeviceToDevice);	
        error_f();
	cublas.sc_prod_vec(vec_o,vec_i,sc,sA,acc);
	error=hipDeviceSynchronize();
	error_f();
}
else
	{fprintf(stderr,"Not implemented\n");exit(-1);}

}


//MATRIX SCALAR OPERATORS

//Scalar product/sum Matrix
//op{0,1}-> 0 is add and 1 is product
//acc{0,1}-> accumulate (1) or set (0)
void gpu_env::scMat(float* mat_o,float* mat_i, tensor_gpu_specs* sA, float sc ,int op,int acc)
{
//gpuI has info about GPU for kernel dimensions

if (op==0)
{
	dim3 dimBlock(sA->row);//maybe move this to declaration
	dim3 dimGrid(sA->col);
	long int ops = sA->row*sA->col;
	//add -> no blas operation (we could use same as product buth we should fill B in gpu and that is shit. Home made kernel)
	sc_add_mat<<<dimBlock,dimGrid>>>(mat_o,mat_i,sc,ops,acc);

	error_f();
	error=hipDeviceSynchronize();
	error_f();
}
else if (op==1)
{
	//prod -> use same as matrix + matrix just matrix B is 0
	
	cublas.sc_prod_mat(mat_o,mat_i,sc,sA,acc);
	error=hipDeviceSynchronize();
	error_f();
}
else
	{fprintf(stderr,"Not implemented\n");exit(-1);}

}

///////////////////////////////////////////
///////////ELEMENT WISE OPERATOR///////////
///////////////////////////////////////////


//(sum or product)
// op{0,1}->0 is add and 1 is product
// acc{0,1}->whether to accumulate result in C (1) or set (0)
//tA and tB are for transposing matrix
//sca and scb only available when sum is done. Not contemplate for el wise product as it is the same as making the product and then scalar dot product. We do not save memory performing this operation in the same operation as in the sum(where we first make a new float* with the result of scalar dot matrix and then sum)
void gpu_env::mat_elwise_mat(float* A, float* B, float* C,tensor_gpu_specs* sA,tensor_gpu_specs* sB,tensor_gpu_specs* sC,int op, int acc, int trA, int trB,float sca, float scb)
{
 
  if (op==0)
  {
	//transpose is done in cublas routine. We have cublas routine for this
	if (acc==0)
	{//not accumulateA
	  cublas.mat_ewsum_mat(A,B,C,sA,sB,trA,trB,sca,scb);	
	}
	else if (acc==1)
	{//accumulate

	 float* aux=makeTensor(sA->row,sA->col,1,1);

         cublas.mat_ewsum_mat(A,B,aux,sA,sB,trA,trB,sca,scb);
	 cublas.mat_ewsum_mat(C,aux,C,sC,sC,0,0);
         destroyTensor(aux);
	}
  }
  else if(op==1) 
  {
	float* tA;
	float* tB;

	if (trA==1)
	{
	   //transpose A
	   tA=makeTensor(sA->row,sA->col,1,1);
   	   cublas.mat_transp(tA,A,sA);
       	   error=hipDeviceSynchronize();
	   error_f();

        }
	if (trB==1)
	{
	   //transpose B
	   tB=makeTensor(sB->row,sB->col,1,1);
   	   cublas.mat_transp(tB,B,sB);
       	   error=hipDeviceSynchronize();
	   error_f();

        }
	dim3 dimBlock(sA->col);//maybe move this to declaration
	dim3 dimGrid(sA->row);
	long int ops = sA->row*sA->col;
        

	if (trA==1 && trB==1)
        {
           mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,tA,tB,acc,ops);
           destroyTensor(tA);
           destroyTensor(tB);
        }
	if (trA==1 && trB==0)
        {
          mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,tA,B,acc,ops);
          destroyTensor(tA);
        }
	if (trA==0 && trB==1){mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,A,tB,acc,ops);destroyTensor(tB);}
	if (trA==0 && trB==0){

            mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,A,B,acc,ops);

         }

	error_f();
	error=hipDeviceSynchronize();
	error_f();


  }
  else
      {fprintf(stderr,"Not implemented\n");exit(-1);}
}

//compare to tensor element by element
int gpu_env::tensor_equal(float* A, float* B,tensor_gpu_specs* sA)
{
  dim3 dimBlock(sA->col);//maybe move this to declaration
  dim3 dimGrid(sA->row);
  long int ops = sA->row*sA->col;

  kern_tensor_equal<<<dimBlock,dimGrid>>>(A,B,ops);
  error=hipDeviceSynchronize();
  if (error==hipErrorLaunchFailure)
	return 0;
  else if(error!=hipSuccess)
	{fprintf(stderr,"Error launching gpu kernel: %s",hipGetErrorString(error));
	exit(-1);}
  else
	return 1;

}

//elwise operator matrix vector
//op=0 is sum and 1 is prod
void gpu_env::mat_elwise_vec(float* mat, float* vec, tensor_gpu_specs* sA,int op)
{
  dim3 dimBlock(sA->col);
  dim3 dimGrid(sA->row);
  long int ops = sA->row*sA->col;

  mat_ewcol_vec<<<dimBlock,dimGrid>>>(mat,vec,ops,sA->col,op);

  error_f();
  error=hipDeviceSynchronize();
  error_f();

}

//elwise operator vector vector

void gpu_env::vec_elwise_vec(float* A, float* B, float* C,tensor_gpu_specs* sA,int op, int acc,float sca, float scb)
{
 //trasposition is not contemplated. This is real programming not grandfather programming. 
  if (op==0)//sum
  {
    cublas.vec_sum_vec(C,B,A,sA,acc,sca,scb);
    error=hipDeviceSynchronize();
    error_f();
 
  }
  else if(op==1) //product
  {
	
      {fprintf(stderr,"Not implemented\n");exit(-1);}
  }
  else
      {fprintf(stderr,"Not implemented\n");exit(-1);}
}


///////////////////////////////////////////////
////////////SELF/REDUCTION OPERATOR////////////
///////////////////////////////////////////////
//This operators should be programed manually using reduction
//techniques. For the moment we can use wonderfull thurst library
//or cuBlas. For performance self-programmed or cuBlas is better.
//We use cannonical float* whenever we can (always looking for avoiding
//warping divergence )
float* gpu_env::col_sum(float* A, tensor_gpu_specs* sA)
{
  dim3 dimBlock(sA->col);
  dim3 dimGrid(1);
  long int ops = sA->col;

  float* O = makeTensor(sA->col);
  kernel_col_sum<<<dimBlock,dimGrid>>>(A,O,(int)sA->row,(int)sA->col,ops);
 
  error_f();
  error=hipDeviceSynchronize();
  error_f();
  return O;
}

int gpu_env::row_max(float* A, tensor_gpu_specs* sA,int ind)
{
//ind is the row index;
//use thrurst for this in future or cubla. As we copy to thurst we only copy the row we operate.
  if (ind > sA->row)
	{fprintf(stderr,"Error. Row not available\n");exit(-1);}
  hipblasStatus_t error_cb;
 // printf ("===Indice pasado %d",ind);
  float *Aaux=A+ind*sA->col;
  int result;
  error_cb = hipblasIsamax(p_cublas,sA->col,Aaux,1,&result);
  if (error_cb!=HIPBLAS_STATUS_SUCCESS)
	{fprintf(stderr,"Error in cublas execution\n");exit(-1);}

//printf("====columns routine %d \n",sA->col);
//printf("====cublas routine %d \n",result);
   
   return result;
}

//////////////////////////////////////////////
///////////////RANDOM GENERATOR///////////////
//////////////////////////////////////////////
/*use host api -> when we need to save generated values. Easier to implement but poor performance:
	-on one side we store values on GPU so waste memory
        -on other side we cannot use generated values online->we loose time accesing memory for reading stored buffers

*/
void gpu_env::random_number_host_binary(float* rand_vec,tensor_gpu_specs* sp,float p)
{

  if(rand_vec==NULL)
     {fprintf(stderr,"Error. Allocate gpu Rand vector\n");exit(-1);}

  rand_error=hiprandGenerateUniform(random_generator,rand_vec,sp->row*sp->col*sp->batch*sp->featureMap);
  error_rand();
  error=hipDeviceSynchronize();
  error_f();

  dim3 dimBlock(sp->col);
  dim3 dimGrid(sp->row);
  long int ops = sp->col*sp->row;


  drop_mask<<<dimBlock,dimGrid>>>(rand_vec,p,ops);
  error_f(); 
  error=hipDeviceSynchronize();
  error_f();

}


void gpu_env::random_number_host_gaussian(float* rand_vec,tensor_gpu_specs* sp,float mean,float std)
{

 rand_error=hiprandGenerateNormal (random_generator, rand_vec, sp->row*sp->col*sp->batch*sp->featureMap,   mean,   std );

  error_rand();
  error=hipDeviceSynchronize();
  error_f();

}

void gpu_env::add_noise(float* vec, float* rand_vec, float noiser, tensor_gpu_specs* sp)
{
  float* mask = makeTensor(sp->row,sp->col);
  rand_error=hiprandGenerateUniform(random_generator,mask,sp->row*sp->col*sp->batch*sp->featureMap);
  error_rand();
  error=hipDeviceSynchronize();
  error_f();
  dim3 dimBlock(sp->col);
  dim3 dimGrid(sp->row);
  long int ops = sp->col*sp->row;

  add_noise_with_mask<<<dimBlock,dimGrid>>>(vec,rand_vec,mask,noiser,ops);
  error_f(); 
  error=hipDeviceSynchronize();
  error_f();
  destroyTensor(mask);


}

/*
void gpu_env::random_number_device_binary(float* rand_vec,tensor_gpu_specs* sp,float p)
{
  if(rand_vec==NULL)
     {fprintf(stderr,"Error. Allocate gpu Rand vector\n");exit(-1);}

  //for avoid to kernel launch(one for generating and one for rounding we do in one. However we take care of actual state)
  rand_error=hiprandGenerateUniform(random_generator,rand_vec,sp->row*sp->col*sp->batch*sp->featureMap);
  error_rand();
  error=hipDeviceSynchronize();
  error_f();
  

}
*/



