#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

#include "../utils.h"
#include "../types.h"
#include "execution.h"
#include "gpu_env.h"
#include "gpu_kernels.h"

#define max_gpu_threads 10000000000

/////////////////CURAND ERROR/////////////////////////


static const char *_curandGetErrorEnum(hiprandStatus_t error)
{
    switch (error)
    {
        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";
            break;
        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";
            break;
        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";
            break;

        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";
            break;

        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";
            break;

        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";
            break;

        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";
            break;

        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
            break;
        default:
	    fprintf(stderr,"Not all hiprand errors here\n");
	    exit(-1);
    }

}

/////////////////////////////////////////////////////////////



gpu_env::gpu_env()
{

}
//error function for random numbers
void gpu_env::error_rand()
{
  if(rand_error!=HIPRAND_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuda random execution getting: %s\n",	_curandGetErrorEnum(rand_error));
     exit(-1);
  }

}
//error function for check internally
void gpu_env::error_f()
{
  if ( error!=hipSuccess)
  {
     fprintf(stderr,"\nProblem in cuda execution getting: %s\n",	hipGetErrorString(error));
     exit(-1);
  }

}

//////////////////////////////////////////
/////////////Memory Transfer//////////////
//////////////////////////////////////////

//JUAN_FIX: change for cuda malloc pitch
//make tensor in GPU and alloc memory
float* gpu_env::makeTensor( int a, int b, int c, int d)
{
  float* devicePointer;
  error=hipMalloc((void**)&devicePointer,a*b*c*d*sizeof(float));
  error_f();
  return devicePointer;
}
//destroy tensor
void gpu_env::destroyTensor(float* p)
{
  error=hipFree(p);
  error_f();
}
//transfer scalar array
void gpu_env::set_sc(float* gpu, float sc, tensor_gpu_specs* sp)
{
  if (gpu==NULL)
    {fprintf(stderr,"Correctly allocate tensor in GPU and CPU\n");exit(-1);}

   //use Memset if int value, for float we need a kernel
     //error=hipMemset(p,sc,size);
   dim3 dimBlock(sp->col);
   dim3 dimGrid(sp->row);
   long int ops=sp->batch*sp->row*sp->col*sp->featureMap;

   tensor_set_value<<<dimBlock,dimGrid>>>(gpu,sc,ops);
   error_f();
   error=hipDeviceSynchronize();
   error_f();
}
//JUAN_FIX:implement function which stack all the device pointer to dealloc when error happens.
//transfer array to array
void gpu_env::copy_data(float* cpu, float* gpu,tr_type t,size_t size)
{
  if (cpu==NULL || gpu==NULL)
	{fprintf(stderr,"Correctly allocate tensor in GPU and CPU\n");exit(-1);}

  if (t==TOGPU)
    error=hipMemcpy(gpu,cpu,size,hipMemcpyHostToDevice);
  else if (t==FROMGPU)
{
    error=hipMemcpy(cpu,gpu,size,hipMemcpyDeviceToHost);
}
  else if (t==GPU)
    error=hipMemcpy(cpu,gpu,size,hipMemcpyDeviceToDevice);
  else
    {fprintf(stderr,"NOT IMPLEMENTED");exit(-1);} 

  error_f();
  error=hipDeviceSynchronize();
  error_f();
}

/////////////////////////////////////
/////////Neural Networks/////////////
/////////////////////////////////////

//activation functions
void gpu_env::activation(float* E,float* N, int f,tensor_gpu_specs* sp)
{
if (E==NULL || N==NULL)
	{fprintf(stderr,"Fill data in pointer.\n");exit(-1);}
dim3 dimBlock;
dim3 dimGrid;
if (f==ACT_SOF)
{
 dimBlock.x=sp->row;
 dimGrid.x=1;
}
else
{
 dimBlock.x=sp->col;
 dimGrid.x=sp->row;
}
long int ops = sp->col*sp->row;
long int sample_dim=sp->col;

double alfa=1;
float* auxE=NULL;
switch(f)
{
  case ACT_RLU:
	ReLu<<<dimBlock,dimGrid>>>(E,N,ops);
	break;
  case ACT_ELU:
	ELU<<<dimBlock,dimGrid>>>(E,N,alfa,ops);
	break;
  case ACT_SOF:
       {
        ops=sp->row;
        auxE = makeTensor(sp->col,sp->row);
        set_sc(auxE, 0.0, sp);
	Softmax<<<dimBlock,dimGrid>>>(E,N,auxE,sample_dim,ops);
	break;
       }
  case ACT_SIG:
	Sigmoid<<<dimBlock,dimGrid>>>(E,N,ops);
	break;
  case ACT_LIN:
        //N=E;
        error = hipMemcpy(N,E,sp->batch*sp->featureMap*sp->col*sp->row*sizeof(float),hipMemcpyDeviceToDevice);
        break;
  default:
	fprintf(stderr,"Activation function not implemented\n");
	exit(-1);
}
error_f();
error=hipDeviceSynchronize();
error_f();
if (auxE!=NULL)
	destroyTensor(auxE);
}
//derivative of activation functions
void gpu_env::dactivation(float* E, float* N,float* D, int f,tensor_gpu_specs* sp)
{
if (E==NULL || N==NULL || D==NULL)
	{fprintf(stderr,"Fill data in pointer.\n");exit(-1);}
dim3 dimBlock(sp->col);
dim3 dimGrid(sp->row);
long int ops = sp->row*sp->col;
long int sample_dim=sp->col*sp->row;

switch(f)
{
  case ACT_RLU:
	dReLu<<<dimBlock,dimGrid>>>(E,D,ops);
	break;
  case ACT_ELU:
	dELU<<<dimBlock,dimGrid>>>(E,N,D,ops);
	break;
  case ACT_SIG:
	dSigmoid<<<dimBlock,dimGrid>>>(E,D,ops);
	break;
  case ACT_LIN:
       set_sc(D,1.0,sp);      
       break;
  default:
	fprintf(stderr,"Activation function not implemented\n");
	exit(-1);
}
error_f();
error=hipDeviceSynchronize();
error_f();

}


//loss functions
void gpu_env::compute_loss(float* T, float* N,loss_type t,tensor_gpu_specs* gsp,double* ent, double* cerr)
{
dim3 dimBlock(gsp->row);
dim3 dimGrid(1);
long int ops = gsp->row;

switch(t)
{
  case CE:
       {
       float* max_row=makeTensor(gsp->row);//store it from device but use in gpu
       int* cerr_g;
       error=hipMalloc((void**)&cerr_g,sizeof(int));
       error_f();
       hipMemset(cerr_g, 0, sizeof(int));
       error_f();
       MC_loss<<<dimBlock,dimGrid>>>(T,N,max_row,gsp->col,ops,cerr_g);
       error_f();
       hipDeviceSynchronize(); 

       error_f();
       float* CE_vec;
       error=hipMalloc((void**)&CE_vec,sizeof(float)*gsp->row*gsp->col);
       error_f();
       CE_loss<<<dimBlock,dimGrid>>>(N,max_row,CE_vec,gsp->col,ops);
       hipDeviceSynchronize(); 
       error_f();
       float* result_ce;
       error=hipMalloc((void**)&result_ce,sizeof(float));
       error_f(); 
       reduce_array_sum<<<dimGrid,dimBlock,ops*sizeof(float)>>>(CE_vec,ops, gsp->col,result_ce);
       hipDeviceSynchronize(); 
       error_f();
       float aux1;
       error=hipMemcpy(&aux1,result_ce,sizeof(float),hipMemcpyDeviceToHost);
       error_f();
       *ent=(double)aux1;
       int aux=0;

       error=hipMemcpy(&aux,cerr_g,sizeof(int),hipMemcpyDeviceToHost);
       *cerr=(double)aux;
       destroyTensor(max_row);
       break;
       }
  case SSE:
	fprintf(stderr,"GPU COST NOT IMPLEMENTED\n");
	exit(-1);
	break;
  default:
	fprintf(stderr,"GPU COST NOT IMPLEMENTED\n");
	exit(-1);

}

}


/////////////////////////////////////////
///////////////gpu parsing///////////////
////////////////////////////////////////
void gpu_env::gpu_info(int selected_gpu)
{

int nDevices;
hipGetDeviceCount(&nDevices);

if (selected_gpu>nDevices)
 {
  fprintf(stderr,"Error. GPU %d not available. Number of available GPU is %d. Further information running nvidia-smi\n",selected_gpu,nDevices);
  exit(-1);	
 }

fprintf(stderr,"Selecting GPU device %d\n",selected_gpu);
hipSetDevice(selected_gpu);

hipDeviceProp_t prop;
hipGetDeviceProperties(&prop,selected_gpu);

fprintf(stderr,"Layers is running on GPU %s\n",prop.name);

gpuI.warpSize=prop.warpSize;
gpuI.maxThreadPerBlock=prop.maxThreadsPerBlock;

}


///////////////////////////////////////////
///////////////Linear Albegra//////////////
///////////////////////////////////////////

//MATRIX TO MATRIX OPERATIONS

//matrix operator
//Perform matrix produc c=a*b
//float pointer are matrix
//tensor_gpu_spcs matrix specifications
//tA and tB are for transposing matrix
// acc{0,1}->whether to accumulate result in C (1) or set (0)
void gpu_env::matMul(float* a,float* b,float* c, tensor_gpu_specs* sA,tensor_gpu_specs* sB, tensor_gpu_specs* sC, int acc, int tA, int tB)
{
//for the moment only available cublas for operation
	cublas.matrixProduct(a,b,c,sA,sB,sC,acc,tA,tB);
	error=hipDeviceSynchronize();
	error_f();

}

//SCALAR VECTOR OPERATOR
void gpu_env::scVec(float* vec_o,float* vec_i, tensor_gpu_specs* sA, float sc ,int op,int acc)
{
//gpuI has info about GPU for kernel dimensions

if (op==0)
{
	dim3 dimBlock(sA->row);//maybe move this to declaration
	dim3 dimGrid(sA->col);
	long int ops = sA->row*sA->col;
	//add -> no blas operation (we could use same as product buth we should fill B in gpu and that is shit. Home made kernel)
	sc_add_mat<<<dimBlock,dimGrid>>>(vec_o,vec_i,sc,ops,acc);//could be same as matrix scalar operator because we treat as float*

	error_f();
	error=hipDeviceSynchronize();
	error_f();
}
else if (op==1)
{
	//prod -> use same as matrix + matrix just matrix B is 0
        if (vec_o==NULL || vec_i==NULL)
		{printf("Error\n");exit(-1);} 
	error=hipMemcpy(vec_o,vec_i,sA->row*sA->col*sizeof(float),hipMemcpyDeviceToDevice);	
        error_f();
	cublas.sc_prod_vec(vec_o,vec_i,sc,sA,acc);
	error=hipDeviceSynchronize();
	error_f();
}
else
	{fprintf(stderr,"Not implemented\n");exit(-1);}

}


//MATRIX SCALAR OPERATORS

//Scalar product/sum Matrix
//op{0,1}-> 0 is add and 1 is product
//acc{0,1}-> accumulate (1) or set (0)
void gpu_env::scMat(float* mat_o,float* mat_i, tensor_gpu_specs* sA, float sc ,int op,int acc)
{
//gpuI has info about GPU for kernel dimensions

if (op==0)
{
	dim3 dimBlock(sA->row);//maybe move this to declaration
	dim3 dimGrid(sA->col);
	long int ops = sA->row*sA->col;
	//add -> no blas operation (we could use same as product buth we should fill B in gpu and that is shit. Home made kernel)
	sc_add_mat<<<dimBlock,dimGrid>>>(mat_o,mat_i,sc,ops,acc);

	error_f();
	error=hipDeviceSynchronize();
	error_f();
}
else if (op==1)
{
	//prod -> use same as matrix + matrix just matrix B is 0
	
	cublas.sc_prod_mat(mat_o,mat_i,sc,sA,acc);
	error=hipDeviceSynchronize();
	error_f();
}
else
	{fprintf(stderr,"Not implemented\n");exit(-1);}

}

///////////////////////////////////////////
///////////ELEMENT WISE OPERATOR///////////
///////////////////////////////////////////
//(sum or product)
// op{0,1,2}->0 is add and 1 is product 2 is sqrt
//Division can be performed by multiplying by inverse and subtraction can be done as a sum using scb and sca correctly (with -)
// acc{0,1}->whether to accumulate result in C (1) or set (0)
//tA and tB are for transposing matrix
//sca and scb only available when sum is done. Not contemplate for el wise product as it is the same as making the product and then scalar dot product. We do not save memory performing this operation in the same operation as in the sum(where we first make a new float* with the result of scalar dot matrix and then sum)
void gpu_env::mat_elwise_mat(float* A, float* B, float* C,tensor_gpu_specs* sA,tensor_gpu_specs* sB,tensor_gpu_specs* sC,int op, int acc, int trA, int trB,float sca, float scb)
{
 
  if (op==0)
  {
	//transpose is done in cublas routine. We have cublas routine for this
	if (acc==0)
	{//not accumulateA
	  cublas.mat_ewsum_mat(A,B,C,sA,sB,trA,trB,sca,scb);	
	}
	else if (acc==1)
	{//accumulate

	 float* aux=makeTensor(sA->row,sA->col,1,1);
         set_sc(aux,0.0, sA);
         cublas.mat_ewsum_mat(A,B,aux,sA,sB,trA,trB,sca,scb);
	 cublas.mat_ewsum_mat(C,aux,C,sC,sC,0,0);
         destroyTensor(aux);
	}
  }
  else if(op==1) 
  {
	float* tA;
	float* tB;

	if (trA==1)
	{
	   //transpose A
	   tA=makeTensor(sA->row,sA->col,1,1);
           set_sc(tA,0.0, sA);
   	   cublas.mat_transp(tA,A,sA);
       	   error=hipDeviceSynchronize();
	   error_f();

        }
	if (trB==1)
	{
	   //transpose B
	   tB=makeTensor(sB->row,sB->col,1,1);
           set_sc(tB,0.0, sB);
   	   cublas.mat_transp(tB,B,sB);
       	   error=hipDeviceSynchronize();
	   error_f();

        }
	dim3 dimBlock(sA->col);//maybe move this to declaration
	dim3 dimGrid(sA->row);
	long int ops = sA->row*sA->col;
        

	if (trA==1 && trB==1)
        {
           mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,tA,tB,acc,ops);
        }
	if (trA==1 && trB==0)
        {
          mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,tA,B,acc,ops);
        }
	if (trA==0 && trB==1){mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,A,tB,acc,ops);}
	if (trA==0 && trB==0){

            mat_ewprod_mat<<<dimBlock,dimGrid>>>(C,A,B,acc,ops);

         }

	error_f();
	error=hipDeviceSynchronize();
	error_f();
	if (trA)
		destroyTensor(tA);
	if (trB)
		destroyTensor(tB);
	error_f(); 

  }
  else
      {fprintf(stderr,"Not implemented\n");exit(-1);}
}

//compare to tensor element by element
int gpu_env::tensor_equal(float* A, float* B,tensor_gpu_specs* sA)
{
  dim3 dimBlock(sA->col);//maybe move this to declaration
  dim3 dimGrid(sA->row);
  long int ops = sA->row*sA->col;

  kern_tensor_equal<<<dimBlock,dimGrid>>>(A,B,ops);
  error=hipDeviceSynchronize();
  if (error==hipErrorLaunchFailure)
	return 0;
  else if(error!=hipSuccess)
	{fprintf(stderr,"Error launching gpu kernel: %s",hipGetErrorString(error));
	exit(-1);}
  else
	return 1;

}

//elwise operator matrix vector
//op=0 is sum and 1 is prod
void gpu_env::mat_elwise_vec(float* mat_o,float* mat, float* vec, tensor_gpu_specs* sA,int op,int acc,float sca, float scb, int rdim)
{
//op 0 is sum
//op 1 is multiplication
//op 2 is division
if (rdim==1)
{
  dim3 dimBlock(sA->col);
  dim3 dimGrid(sA->row);
  long int ops = sA->row*sA->col;

  mat_ewcol_vec<<<dimBlock,dimGrid>>>(mat_o,mat,vec,ops,sA->col,op,acc,sca, scb);
}
else
{
  dim3 dimBlock(sA->col);
  dim3 dimGrid(sA->row);
  long int ops = sA->row*sA->col;

  mat_ewrow_vec<<<dimBlock,dimGrid>>>(mat_o,mat,vec,ops,sA->row,op,acc,sca, scb);
}

  error_f();
  error=hipDeviceSynchronize();
  error_f();
}

//elwise operator vector vector

void gpu_env::vec_elwise_vec(float* A, float* B, float* C,tensor_gpu_specs* sA,int op, int acc,float sca, float scb)
{
 //trasposition is not contemplated. This is real programming not grandfather programming. 
  if (op==0)//sum
  {
    cublas.vec_sum_vec(C,B,A,sA,acc,sca,scb);
    error=hipDeviceSynchronize();
    error_f();
 
  }
  else if(op==1) //product
  {
	
      {fprintf(stderr,"Not implemented\n");exit(-1);}
  }
  else
      {fprintf(stderr,"Not implemented\n");exit(-1);}
}
///////////////////////////////////////////////
/////////////////INPLACE OPERATOR//////////////
///////////////////////////////////////////////
void gpu_env::mat_inplace_mat(float* o, float* i,tensor_gpu_specs* si,int op, int acc)
{
  dim3 dimBlock(si->row);
  dim3 dimGrid(si->col);
  if (op==0)//sqrt
  {
    
  tensor_sqrt<<<dimGrid,dimBlock>>>(o,i, acc,si->col*si->row);
  }
  else
  {fprintf(stderr,"Not implemented mat_inplace_mat\n");exit(-1);}    
}


///////////////////////////////////////////////
///////////////REDUCTION OPERATOR//////////////
///////////////////////////////////////////////
//This operators should be programed manually using reduction
//techniques. For the moment we can use wonderfull thurst library
//or cuBlas. For performance self-programmed or cuBlas is better.
//We use cannonical float* whenever we can (always looking for avoiding
//warping divergence )
void gpu_env::sum_abs(float* p,tensor_gpu_specs* gsp,float* acc)
{
hipblasStatus_t error_cb=hipblasSasum(p_cublas,gsp->row*gsp->col,p,1,acc);
if (error_cb!=HIPBLAS_STATUS_SUCCESS)
	fprintf(stderr,"Error calling sum_abs with hipblasSasum routine\n");

}

void gpu_env::reduce_operator(float* p,tensor_gpu_specs* gsp,float* acc)
{
  float* sum_aux;
  error = hipMalloc((void**)&sum_aux,sizeof(float));
  error_f();
  dim3 dimGrid(1);
  dim3 dimBlock(gsp->row);
  long int ops = gsp->row;
  reduce_array_sum<<<dimGrid,dimBlock,ops*sizeof(float)>>>(p,ops, gsp->col,sum_aux);
  hipDeviceSynchronize();
  error_f();
  copy_data(acc,sum_aux,FROMGPU,sizeof(float));
 

}

void gpu_env::row_sum(float* A, tensor_gpu_specs* sA,float* B)
{
  dim3 dimBlock(sA->col);
  dim3 dimGrid(1);
  long int ops = sA->row;

  tensor_set_value<<<dimBlock,dimGrid>>>(B,0.0,sA->row);
  error_f();
  error=hipDeviceSynchronize();
  error_f();

  kernel_row_sum<<<dimBlock,dimGrid>>>(A,B,(int)sA->row,(int)sA->col,ops);

  error_f();
  error=hipDeviceSynchronize();
  error_f();
}


void gpu_env::col_sum(float* A, tensor_gpu_specs* sA,float* B)
{
  dim3 dimBlock(sA->col);
  dim3 dimGrid(1);
  long int ops = sA->col;

  tensor_set_value<<<dimBlock,dimGrid>>>(B,0.0,sA->col);
  error_f();
  error=hipDeviceSynchronize();
  error_f();

  kernel_col_sum<<<dimBlock,dimGrid>>>(A,B,(int)sA->row,(int)sA->col,ops);
 
  error_f();
  error=hipDeviceSynchronize();
  error_f();
  
}

int gpu_env::row_max(float* A, tensor_gpu_specs* sA,int ind)
{
//ind is the row index;
//use thrurst for this in future or cubla. As we copy to thurst we only copy the row we operate.
  if (ind > sA->row)
	{fprintf(stderr,"Error. Row not available\n");exit(-1);}
  hipblasStatus_t error_cb;
 // printf ("===Indice pasado %d",ind);
  float *Aaux=A+ind*sA->col;
  int result;
  error_cb = hipblasIsamax(p_cublas,sA->col,Aaux,1,&result);
  if (error_cb!=HIPBLAS_STATUS_SUCCESS)
	{fprintf(stderr,"Error in cublas execution\n");exit(-1);}

//printf("====columns routine %d \n",sA->col);
//printf("====cublas routine %d \n",result);
   hipDeviceSynchronize();
   error_f();
   return result;
} 

//////////////////////////////////////////////
///////////////RANDOM GENERATOR///////////////
//////////////////////////////////////////////
/*use host api -> when we need to save generated values. Easier to implement but poor performance:
	-on one side we store values on GPU so waste memory
        -on other side we cannot use generated values online->we loose time accesing memory for reading stored buffers

*/
void gpu_env::random_number_host_binary(float* rand_vec,tensor_gpu_specs* sp,float p)
{

  if(rand_vec==NULL)
     {fprintf(stderr,"Error. Allocate gpu Rand vector\n");exit(-1);}

  rand_error=hiprandGenerateUniform(random_generator,rand_vec,sp->row*sp->col*sp->batch*sp->featureMap);
  error_rand();
  error=hipDeviceSynchronize();
  error_f();

  dim3 dimBlock(sp->col);
  dim3 dimGrid(sp->row);
  long int ops = sp->col*sp->row;


  drop_mask<<<dimBlock,dimGrid>>>(rand_vec,p,ops);
  error_f(); 
  error=hipDeviceSynchronize();
  error_f();

}


void gpu_env::random_number_host_gaussian(float* rand_vec,tensor_gpu_specs* sp,float mean,float std)
{

 rand_error=hiprandGenerateNormal (random_generator, rand_vec, sp->row*sp->col*sp->batch*sp->featureMap,   mean,   std );

  error_rand();
  error=hipDeviceSynchronize();
  error_f();

}

void gpu_env::add_noise(float* vec, float* rand_vec, float noiser, tensor_gpu_specs* sp)
{
  float* mask = makeTensor(sp->row,sp->col);
  rand_error=hiprandGenerateUniform(random_generator,mask,sp->row*sp->col*sp->batch*sp->featureMap);
  error_rand();
  error=hipDeviceSynchronize();
  error_f();
  dim3 dimBlock(sp->col);
  dim3 dimGrid(sp->row);
  long int ops = sp->col*sp->row;

  add_noise_with_mask<<<dimBlock,dimGrid>>>(vec,rand_vec,mask,noiser,ops);
  error_f(); 
  error=hipDeviceSynchronize();
  error_f();
  destroyTensor(mask);


}

/*
void gpu_env::random_number_device_binary(float* rand_vec,tensor_gpu_specs* sp,float p)
{
  if(rand_vec==NULL)
     {fprintf(stderr,"Error. Allocate gpu Rand vector\n");exit(-1);}

  //for avoid to kernel launch(one for generating and one for rounding we do in one. However we take care of actual state)
  rand_error=hiprandGenerateUniform(random_generator,rand_vec,sp->row*sp->col*sp->batch*sp->featureMap);
  error_rand();
  error=hipDeviceSynchronize();
  error_f();
  

}
*/



