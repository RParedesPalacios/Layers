#include "gpu_utils.h"
#include <hipblas.h>
#include <string.h>
#include <stdio.h>
#include <hiprand.h>

 
bool useCPU=0;

//For controlling cublas routines
bool isYes=0; //for controlling cublas activation

//For controlling tensor operations over GPU
hipblasHandle_t p_cublas;
hipblasStatus_t status;
gpu_specs gpuI;
gpu_env gpu_tensor_op;


//for controlling random generator
long int gpu_seed=10;
hiprandGenerator_t random_generator;
hiprandStatus_t rand_error;


 
void cublasInit()
{

  status=hipblasCreate(&p_cublas);
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas execution getting: NOT IMPLEMENTED \n");
     exit(1);

  }

  isYes=1;

/*  status=hipblasSetPointerMode(p_cublas,HIPBLAS_POINTER_MODE_DEVICE);
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas execution getting: NOT IMPLEMENTED \n");
     exit(1);

  }*/

}

void curandInit()
{

  rand_error=hiprandCreateGenerator(&random_generator,HIPRAND_RNG_PSEUDO_MRG32K3A);
  if (rand_error != HIPRAND_STATUS_SUCCESS)
	{fprintf(stderr,"Error creating random numbers on gpu\n");exit(-1);}

   rand_error=hiprandSetPseudoRandomGeneratorSeed(random_generator,gpu_seed); 
    if (rand_error != HIPRAND_STATUS_SUCCESS)
	{fprintf(stderr,"Error seeting the seed for program\n");exit(-1);}

}





















