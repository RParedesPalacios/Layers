#include "execution.h"
#include <hiprand.h> 
#include <hipblas.h>
#include <string.h>
#include <stdio.h>

//For controlling tensor operations over GPU
hipblasHandle_t p_cublas;//for cublas routines execution
hipblasStatus_t status;//cublas error
bool isYes=0;//for cublas execution

gpu_env gpu_tensor_op;//gpu class routines

//for controlling random generator
hiprandGenerator_t random_generator; //for random routines
hiprandStatus_t rand_error;// random error

gpu_specs gpuI;
long int gpu_seed=1;

//initialice cublas handle
void cublasInit()
{
  int i=0;

  status=hipblasCreate(&p_cublas);
  while ((status!=  HIPBLAS_STATUS_SUCCESS)&&(i<10)) {
    status=hipblasCreate(&p_cublas);
    i++;
    fprintf(stderr,".");
  }
  fprintf(stderr,"\n");
  
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas Create\n");
     exit(1);

  }

  isYes=1;

status = hipblasSetAtomicsMode(p_cublas,HIPBLAS_ATOMICS_NOT_ALLOWED);
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas execution getting: NOT IMPLEMENTED \n");
     exit(1);

  }


/*  status=hipblasSetPointerMode(p_cublas,HIPBLAS_POINTER_MODE_DEVICE);
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas execution getting: NOT IMPLEMENTED \n");
     exit(1);

  }*/

}

//initialize hiprand handle and set seed
void curandInit()
{

  rand_error=hiprandCreateGenerator(&random_generator,HIPRAND_RNG_PSEUDO_MRG32K3A);
  if (rand_error != HIPRAND_STATUS_SUCCESS)
	{fprintf(stderr,"Error creating random numbers on gpu\n");exit(-1);}

   rand_error=hiprandSetPseudoRandomGeneratorSeed(random_generator,gpu_seed); 
    if (rand_error != HIPRAND_STATUS_SUCCESS)
	{fprintf(stderr,"Error seeting the seed for program\n");exit(-1);}

}


